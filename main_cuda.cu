#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <inttypes.h>

#define WS 8             // word size
#define CN 62            // characters number
#define DICT_SIZE 285465 // dictionary size
#define THREADS_NUM 512  // number of threads for each block

#include "../utils/aes.h"
#include "../utils/aes.c"

#ifdef __CUDA_ARCH__
#define CONSTANT __constant__
#else
#define CONSTANT
#endif

__constant__ unsigned char c_target[32]; // constant memory copy of encrypted password

CONSTANT uint8_t key[] = "aaaaaaaaaaaaaaaa";
CONSTANT uint8_t iv[] = "bbbbbbbbbbbbbbbb";
CONSTANT uint8_t str[] = "00000000"; //password
CONSTANT struct AES_ctx ctx;

//Custom implematation of memcmp for cuda
__device__ __host__ int cuda_memcmp(void *s1, void *s2, int n)
{
    unsigned char *p = (unsigned char *)s1;
    unsigned char *q = (unsigned char *)s2;
    if (s1 == s2)
    {
        return 0;
    }
    for (int i = 0; i < n; i++)
    {
        if (p[i] != q[i])
        {
            return -1;
        }
    }
    return 0;
}

__host__ __device__ void encrypt(uint8_t *plain)
{
    AES_init_ctx_iv(&ctx, key, iv);
    AES_CBC_encrypt_buffer(&ctx, plain, 32);
}

__host__ __device__ void decrypt(uint8_t *cipher)
{
    AES_init_ctx_iv(&ctx, key, iv);
    AES_CBC_decrypt_buffer(&ctx, cipher, 32);
}

__host__ __device__ void uint64_to_uint8(uint8_t mesg[], uint64_t num)
{
    for (int i = 0; i < 8; i++)
        mesg[i] = num >> (8 - 1 - i) * 8;
}

// dictionary kernel
__global__ void dict_kernel(unsigned char *dictionary, unsigned char *result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < DICT_SIZE)
    {
        unsigned char *word = (unsigned char *)dictionary[index * 8];
        /*Extend to 32 while content is 8*/
        uint8_t buf[32];
        for (int i = 0; i < 32; i++)
        {
            if (i < 8)
            {
                buf[i] = (uint8_t)word[i];
            }
            else
            {
                buf[i] = 0x00;
            }
        }
        uint8_t *buf_p = (uint8_t *)buf;
        encrypt(buf_p);

        // the thread found the solution
        if (cuda_memcmp(buf_p, c_target, 32) == 0)
        {
            result = (unsigned char *)word;
            return;
        }
    }
}

__global__ void brute_kernel(uint8_t *result, int offset, int *success)
{
    uint64_t word = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (word < 0xFFFFFFFFFFFFFFFF)
    {
        word += 3472328296227680304; // index translation: thread 0 tries "00000000" and so on

        //convert uint64_t word to uint8_t array
        uint8_t *uword = (uint8_t *)malloc(8 * sizeof(uint8_t));
        uint64_to_uint8(uword, word);

        /*Extend to 32 while content is 8*/
        uint8_t buf[32];
        for (int i = 0; i < 32; i++)
        {
            if (i < 8)
            {
                buf[i] = (uint8_t)uword[i];
            }
            else
            {
                buf[i] = 0x00;
            }
        }
        uint8_t *buf_p = (uint8_t *)buf; //

        encrypt(buf_p);
        // to avoid Memory err
        free(uword);
        if (cuda_memcmp(buf_p, c_target, 32) == 0)
        {                                  // the thread found the solution
            result = (unsigned char *)str; //uword;
            *success = 1;
            return;
        }
    }
}

int main(int argc, char **argv)
{

    unsigned char *upassword;
    uint8_t *crypted_target;
    char *curr_word = (char *)malloc(WS * sizeof(char));
    unsigned char *u_curr_word;
    FILE *dictionary;
    // device & host dictionary
    unsigned char h_dictionary[DICT_SIZE][WS];
    unsigned char *d_dictionary;

    // decryption result
    unsigned char *result = NULL;
    unsigned char *d_result = NULL;
    int *success;
    int _success = 0;

    hipError_t err;

    // password to find
    char password[] = "00000000";

    // verify if the user inserted eight characters password
    if ((int)strlen(password) != 8)
    {
        printf("%d\n", (int)strlen(password));
        perror("error: insert an eight characters password");
    }
    printf("target:%s\n", password);

    // conversion and encryption
    upassword = (unsigned char *)(password);
    printf("encoded target:");
    for (int i = 0; i < 8; i++)
    {
        printf("%02x", upassword[i]);
    }
    /*Extend to 32 while content is 8*/
    /*We will use this inside encrypt/decrypt only*/
    uint8_t buf[32];
    for (int i = 0; i < 32; i++)
    {
        if (i < 8)
        {
            buf[i] = (uint8_t)upassword[i];
        }
        else
        {
            buf[i] = 0x00;
        }
    }
    uint8_t *buf_p = (uint8_t *)buf;
    printf("\nencoded buf:");
    for (int i = 0; i < 32; ++i)
    {
        printf("%.2x", buf_p[i]);
    }
    encrypt(buf_p);

    crypted_target = (uint8_t *)buf_p;

    printf("\nencrypted target:");
    for (int i = 0; i < 32; i++)
    {
        printf("%02x", crypted_target[i]);
    }

    // start counting clock cycles
    clock_t start_t = clock();

    puts("\nPhase 1: Try with dictionary");
    puts("opening dictionary...");
    if ((dictionary = fopen("./dictionary.txt", "r")) == NULL)
    {
        perror("error: dictionary not found");
    }
    puts("dictionary opened...");
    puts("");

    // dictionary import and converting
    puts("dictionary import and converting...");
    int i = 0;
    while (!feof(dictionary))
    {
        fscanf(dictionary, "%8s", curr_word);
        u_curr_word = (unsigned char *)curr_word;
        for (int j = 0; j < WS; j++)
        {
            h_dictionary[i][j] = u_curr_word[j];
        }
        i++;
    }

    //closing the file
    fclose(dictionary);
    puts("import/conversion done...");
    puts("");

    // dictionary attack
    // gpu malloc and memset
    puts("\ngpu malloc and memset...");
    hipMalloc((void **)&success, sizeof(int));
    hipMemset(success, 0, sizeof(int));
    err = hipMalloc((void **)&d_result, WS * sizeof(unsigned char));
    if (err != hipSuccess)
    {
        printf("\n %s\n", hipGetErrorString(err));
    }
    hipMemcpyToSymbol(HIP_SYMBOL(c_target), crypted_target, 32 * sizeof(unsigned char));

    err = hipMalloc((void **)&d_dictionary, DICT_SIZE * WS * sizeof(unsigned char));
    if (err != hipSuccess)
    {
        printf("\n %s\n", hipGetErrorString(err));
    }
    puts("malloc and memset done...");
    puts("");

    //gpu memcpy
    puts("gpu memcpy...");
    hipMemcpy(d_dictionary, h_dictionary, DICT_SIZE * WS * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_target), crypted_target, 32 * sizeof(unsigned char));

    puts("gpu memcpy done...");
    puts("");

    // dictionary kernel launch
    puts("dictionary kernel launch...");
    int block_size = DICT_SIZE / THREADS_NUM + 1;
    dict_kernel<<<block_size, THREADS_NUM>>>(d_dictionary, d_result);

    // copying result
    hipMemcpy(result, d_result, WS * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //check if password was found
    if (result != NULL)
    {
        printf("\npassword found:");
        for (int i = 0; i < 8; i++)
        {
            printf("%02x ", result[i]);
        }

        // gpu memory deallocation
        hipFree(d_dictionary);

        // stop counting clock cycles and calculate elapsed time
        clock_t end_t = clock();
        clock_t total_t = (end_t - start_t);
        printf("\nElapsed Time:%.3f seconds\n", (double)total_t / ((double)CLOCKS_PER_SEC));

        return 0;
    }
    else
    {
        puts("password not in dictionary...");
    }

    // gpu memory deallocation
    hipFree(d_dictionary);

    // Phase 2
    puts("\nPhase2: brute force. This may take a long time...");
    unsigned long long brute_size = 0xFFFFFFFFFFFFFFFF;
    unsigned int brute_blocks = 512, brute_threads = 512;

    // a kernel launch processes (brute_blocks * brute_threads) elements
    //compare the crypted_target with possible hashes (2^64 − 1 which equals 18,446,744,073,709,551,615)
    //(ps le temps CUDA peut être amélioré en parallelisant la boucle exterieure)
    //dim3 block(brute_blocks, brute_threads)
    //dim3 grid
    for (int i = 0; i < (brute_size / (brute_blocks * brute_threads)) + 1; i++)
    {
        brute_kernel<<<brute_blocks, brute_threads>>>(d_result, i * (brute_blocks * brute_threads), success);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            break;
        }
        hipMemcpy(&_success, success, sizeof(int), hipMemcpyDeviceToHost);

        /*// copying result
		    err = hipMemcpy(result, d_result, WS*sizeof(unsigned char), hipMemcpyDeviceToHost);
        if ( err != hipSuccess ){
          printf("CUDA Error: %s\n", hipGetErrorString(err));       
        } */

        //check if password was found
        if (_success == 1)
        {
            printf("\nsucess %d\n", _success);
            printf("\npassword found:");
            for (int i = 0; i < 8; i++)
            {
                printf("%02x ", upassword[i]);
            }
            break;
        }
    }

    // stop counting clock cycles and calculate elapsed time
    clock_t end_t = clock();
    clock_t total_t = (end_t - start_t);
    printf("\nElapsed Time:%f seconds\n", (double)total_t / ((double)CLOCKS_PER_SEC));

    hipFree(d_result);
    hipFree(success);

    return 0;
}